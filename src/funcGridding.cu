#include "hip/hip_runtime.h"
/**
 *    @file funcGridding.cu
 *    @author Peijin Zhang
 *    The kernel of gridding function
 *    reference: Holger Rapp 2009-03-11. 
 * (https://github.com/astroumd/miriad/blob/master/src/subs/mapper.for)
 * (https://github.com/astroumd/miriad/blob/master/src/subs/grid.for
 */

 #include <math.h>
 #include <stdio.h>
 #include "funcGridding.cuh"
 #include "hip/hip_runtime.h"
 
 #include "hipfft/hipfft.h"
 #include "hip/hip_complex.h"
 
// cgf: convolutional gridding function

 extern "C" {
	__global__ void testMem(int * a)
	{
		if (threadIdx.x == 0 && blockIdx.x == 0)
		{
			printf("a = %f\n", 1.1);
		}
	}
	
   __global__ void SimpleGridding(float2 * Grd, float2 * bm, \
	 float2 * sf, int * cnt, float * d_u, float * d_v, float * d_re,
	 float * d_im, float * cgf, int WIDTH, int HWIDTH, int NCGF, 
	 int nu, float du, int gcount, int umax, int vmax, 
	 int batch_size_img, int batch_size_vis) {
	 
		// gridding function
	 // Grd: gridded data output
	 // bm:  beam in the gridding, (fft of the dirty beam)
	 // sf:  binary beam in the gridding
	 // cnt: counter of the gridding
	 // d_u: u coordinate of the visibility
	 // d_v: v coordinate of the visibility
	 // d_re: real part of the visibility
	 // d_im: imaginary part of the visibility
	 // nu:  number of u 
	 // du:  size of u pixel
	 // gcount: number of visibilities
	 // umax:  maximum u pixel
	 // vmax:  maximum v pixel
 
	 int iu = blockDim.x * blockIdx.x + threadIdx.x;
	 int iv_block = threadIdx.y; 
	 int u0 = 0.5 * nu; // center of u and v
	 int iv;

	 for (iv=iv_block*batch_size; iv<(iv_block+1)*batch_size; iv++) {
		if (iu >= u0 && iu <= u0 + umax && iv <= u0 + vmax) {
			// consider u>0
			for (int ivis = 0; ivis < gcount; ivis++) {
			float mu = d_u[ivis];
			float mv = d_v[ivis];
			int hflag = 1;
			if (mu < 0) { // for u<0, do conjugate
				hflag = -1;
				mu = -1 * mu;
				mv = -1 * mv;
			}
			float uu, vv; // u, v in pixel space, but still decimal
			uu = mu / du + u0;
			vv = mv / du + u0;

			int cnu = abs(iu - uu), cnv = abs(iv - vv); 
			// distance of the pixel to the visibility
			
			if (cnu < HWIDTH && cnv < HWIDTH) {
				int ind = iv * nu + iu;
				float wgt = cgf[int(round((NCGF-1.0)/WIDTH * cnu + (NCGF-1.0)/2))] *\
								cgf[int(round((NCGF-1.0)/WIDTH * cnv + (NCGF-1.0)/2))];
				Grd[ind].x += wgt * d_re[ivis];
				Grd[ind].y += hflag * wgt * d_im[ivis];
				cnt[ind] += 1;
				bm[ind].x += wgt;
				sf[ind].x = 1;
				sf[ind].y = 1;
			}
			
			// deal with points&pixels close to u=0 boundary
			if (iu - u0 < HWIDTH && mu / du < HWIDTH) {
				int ind = iv * nu + iu;
				mu = -1 * mu;
				mv = -1 * mv;
				uu = mu / du + u0;
				vv = mv / du + u0;
				cnu = abs(iu - uu), cnv = abs(iv - vv);
				if (cnu < HWIDTH && cnv < HWIDTH) {
				float wgt = cgf[int(round(4.6 * cnu + NCGF - 0.5))] * cgf[int(round(4.6 * cnv + NCGF - 0.5))];
				Grd[ind].x += wgt * d_re[ivis];
				Grd[ind].y += -1 * hflag * wgt * d_im[ivis];
				cnt[ind] += 1;
				bm[ind].x += wgt;
				sf[ind].x = 1;
				sf[ind].y = 1;
				}
			}
			}
		}
   }
	}
 }